#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <cstdio>
#include <cmath>

#include "md5.cuh"

__constant__ u8 player_name_prefix[] = {
    'O', 'f', 'f', 'l', 'i', 'n', 'e', 'P', 'l', 'a', 'y', 'e', 'r', ':'
};

constexpr auto player_name_prefix_length = 14;

__constant__ u8 available_chars[] = {
    '0', '1', '2', '3', '4', '5', '6', '7', '8', '9',
    'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p', 'q', 'r', 's', 't', 'u', 'v', 'w',
    'x', 'y', 'z',
    'A', 'B', 'C', 'D', 'E', 'F', 'G', 'H', 'I', 'J', 'K', 'L', 'M', 'N', 'O', 'P', 'Q', 'R', 'S', 'T', 'U', 'V', 'W',
    'X', 'Y', 'Z',
    '_'
};

constexpr auto available_char_length = 63;
constexpr auto available_char_length_pow_2 = 63 * 63;
constexpr auto available_char_length_pow_3 = 63 * 63 * 63;

constexpr auto player_name_max_length = 16;

__device__ void get_next_index(u8* num, const int length)
{
    // base 63 integer
    int carry = 1;

    for (int i = length - 1; i >= 0 && carry > 0; --i)
    {
        num[i] += carry;
        if (num[i] >= 63)
        {
            num[i] = 0;
            carry = 1;
        }
        else
        {
            carry = 0;
        }
    }
}

__device__ void convert_md5_to_u128(const u8 md5[md5_block_size], u64* hi, u64* lo)
{
    *hi = (static_cast<u64>(md5[0]) << 56) | (static_cast<u64>(md5[1]) << 48) |
        (static_cast<u64>(md5[2]) << 40) | (static_cast<u64>(md5[3]) << 32) |
        (static_cast<u64>(md5[4]) << 24) | (static_cast<u64>(md5[5]) << 16) |
        (static_cast<u64>(md5[6]) << 8) | static_cast<u64>(md5[7]);

    *lo = (static_cast<u64>(md5[8]) << 56) | (static_cast<u64>(md5[9]) << 48) |
        (static_cast<u64>(md5[10]) << 40) | (static_cast<u64>(md5[11]) << 32) |
        (static_cast<u64>(md5[12]) << 24) | (static_cast<u64>(md5[13]) << 16) |
        (static_cast<u64>(md5[14]) << 8) | static_cast<u64>(md5[15]);
}

void convert_md5_to_u128_cpu(const u8 md5[md5_block_size], u64* hi, u64* lo)
{
    *hi = (static_cast<u64>(md5[0]) << 56) | (static_cast<u64>(md5[1]) << 48) |
        (static_cast<u64>(md5[2]) << 40) | (static_cast<u64>(md5[3]) << 32) |
        (static_cast<u64>(md5[4]) << 24) | (static_cast<u64>(md5[5]) << 16) |
        (static_cast<u64>(md5[6]) << 8) | static_cast<u64>(md5[7]);

    *lo = (static_cast<u64>(md5[8]) << 56) | (static_cast<u64>(md5[9]) << 48) |
        (static_cast<u64>(md5[10]) << 40) | (static_cast<u64>(md5[11]) << 32) |
        (static_cast<u64>(md5[12]) << 24) | (static_cast<u64>(md5[13]) << 16) |
        (static_cast<u64>(md5[14]) << 8) | static_cast<u64>(md5[15]);
}

__global__ void kernel_md5_hash_player_name(const int length, u8* cuda_indata, u8* cuda_outdata)
{
    u32 thread = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread >= available_char_length_pow_3)
    {
        return;
    }

    // pattern: [prefix][in_traversal_part][byte_a][byte_b][byte_c]
    // we are trying to find the best [in_traversal_part] that gives the smallest MD5 hash (offline uuid)
    // by iterating through all possible [in_traversal_part]
    //
    // prefix: "OfflinePlayer:" 14 bytes
    // in_traversal_part: (length) bytes
    // byte_a, byte_b, byte_c: 1 byte each
    int byte_a_idx = thread / (available_char_length_pow_2);
    int byte_b_idx = (thread % (available_char_length_pow_2)) / available_char_length;
    int byte_c_idx = thread % available_char_length;

    u8 byte_a = available_chars[byte_a_idx];
    u8 byte_b = available_chars[byte_b_idx];
    u8 byte_c = available_chars[byte_c_idx];

    // best result within this thread
    u8 local_best_in[player_name_max_length] = {0};
    u8 local_best_out[md5_block_size] = {0};

    for (unsigned char& i : local_best_in)
    {
        i = UINT8_MAX;
    }
    for (unsigned char& i : local_best_out)
    {
        i = UINT8_MAX;
    }

    // in_traversal_part is a base 63 integer as the index of available_chars
    u8 in_traversal_part[player_name_max_length] = { 0 };

    // iterate through all possible player names with (length + 3) characters
    for (int _ = 0; _ < pow(available_char_length, length); ++_)
    {
        // add 1 to in_traversal_part
        get_next_index(in_traversal_part, length);

        // assemble the MD5 input
        u8 in[player_name_prefix_length + player_name_max_length] = { 0 };
        for (int i = 0; i < player_name_prefix_length; ++i)
        {
            in[i] = player_name_prefix[i];
        }
        for (int i = length - 1; i >= 0; --i)
        {
            in[player_name_prefix_length + i] = available_chars[in_traversal_part[i]];
        }
        in[player_name_prefix_length + length] = byte_a;
        in[player_name_prefix_length + length + 1] = byte_b;
        in[player_name_prefix_length + length + 2] = byte_c;

        // calculate MD5 hash
        u32 inlen = player_name_prefix_length + length + 3;
        u8 out[md5_block_size];

        cuda_md5_ctx ctx;
        cuda_md5_init(&ctx);
        cuda_md5_update(&ctx, in, inlen);
        cuda_md5_final(&ctx, out);

        // compare with the best result within this thread
        // msvc does not support u128, so we have to compare hi and lo separately
        u64 current_out_hi, current_out_lo, local_best_out_hi, local_best_out_lo;

        convert_md5_to_u128(out, &current_out_hi, &current_out_lo);
        convert_md5_to_u128(cuda_outdata, &local_best_out_hi, &local_best_out_lo);

        if (current_out_hi < local_best_out_hi || (current_out_hi == local_best_out_hi && current_out_lo < local_best_out_lo))
        {
            for (int i = 0; i < player_name_max_length; ++i)
            {
                local_best_in[i] = in[player_name_prefix_length + i];
            }
            for (int i = 0; i < md5_block_size; ++i)
            {
                local_best_out[i] = out[i];
            }
        }
    }

    // write the best result within this thread to global memory
    for (int i = 0; i < player_name_max_length; ++i)
    {
        cuda_indata[thread * player_name_max_length + i] = local_best_in[i];
    }
    for (int i = 0; i < md5_block_size; ++i)
    {
        cuda_outdata[thread * md5_block_size + i] = local_best_out[i];
    }
}

int main()
{
    // test 1: n-batch md5 hash
    u8 in[21] = {
        'O', 'f', 'f', 'l', 'i', 'n', 'e', 'P', 'l', 'a', 'y', 'e', 'r', ':', 'C', 'a', 't', 'M', 'e', '0', 'w'
    };
    u8 out[md5_block_size];
    mcm_cuda_md5_hash_batch(in, 21, out, 1);
    for (const u8 i : out)
    {
        printf("%02x", i);
    }
    printf("\n");

    // test 2: 6 chars player name md5 hash
    u8* cuda_indata;
    u8* cuda_outdata;
    hipMalloc(&cuda_indata, available_char_length_pow_3 * player_name_max_length);
    hipMalloc(&cuda_outdata, available_char_length_pow_3 * md5_block_size);

    // 250112 threads (250047 used), 977 blocks
    int thread = 256;
    int block = (available_char_length_pow_3 + thread - 1) / thread;

    kernel_md5_hash_player_name << < block, thread >> > (3, cuda_indata, cuda_outdata);
    hipDeviceSynchronize();

    auto indata = new u8[available_char_length_pow_3 * player_name_max_length];
    auto outdata = new u8[available_char_length_pow_3 * md5_block_size];
    hipMemcpy(indata, cuda_indata, available_char_length_pow_3 * player_name_max_length, hipMemcpyDeviceToHost);
    hipMemcpy(outdata, cuda_outdata, available_char_length_pow_3 * md5_block_size, hipMemcpyDeviceToHost);

    const hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        printf("Error kernel_md5_hash_player_name: %s \n", hipGetErrorString(error));
    }

    hipFree(cuda_indata);
    hipFree(cuda_outdata);

    // find the best player name in the results
    u8 best_in[player_name_max_length] = { 0 };
    u64 best_out_hi = ULLONG_MAX;
    u64 best_out_lo = ULLONG_MAX;

    for (int i = 0; i < available_char_length_pow_3; ++i)
    {
        u64 hi, lo;
        convert_md5_to_u128_cpu(outdata + i * md5_block_size, &hi, &lo);
        if (hi < best_out_hi || (hi == best_out_hi && lo < best_out_lo))
        {
            for (int j = 0; j < player_name_max_length; ++j)
            {
                best_in[j] = indata[i * player_name_max_length + j];
            }
            best_out_hi = hi;
            best_out_lo = lo;
        }
    }

    printf("Best player name: ");
    for (const u8 i : best_in)
    {
        printf("%c", i);
    }
    printf("\nMD5: ");
    printf("%016llx%016llx\n", best_out_hi, best_out_lo);

    return 0;
}
